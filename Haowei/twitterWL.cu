//
// Created by depaulsmiller on 1/15/21.
//

#include <unistd.h>
#include "helper.cuh"
#include <algorithm>
#include <boost/property_tree/ptree.hpp>
#include <boost/property_tree/json_parser.hpp>
#include <dlfcn.h>
#include <fstream>
#include <iostream>
#include <string>
#include <sstream>

namespace pt = boost::property_tree;
using BatchWrapper = std::vector<RequestWrapper<unsigned long long, data_t *>>;
//#ifdef MODEL_CHANGE
using Model = kvgpu::AnalyticalModel<unsigned long long>;
//#else
//using Model = kvgpu::SimplModel<unsigned long long>;
//#endif
using RB = std::shared_ptr<Communication>;
using namespace std;
int RATIO_OF_READS = 95;
int totalBatches = 10000;
int BATCHSIZE = 512;
int KEY_SIZE = 8;
int NUM_THREADS = 4;//std::thread::hardware_concurrency() - 10;

void usage(char *command);
int count_workload(ifstream &inFile);
std::vector<BatchWrapper> getPopulationBatches(ifstream &inFile, unsigned seed, unsigned int nums);
std::vector<RequestWrapper<unsigned long long, data_t *>> generateWorkloadBatch(ifstream &inFile, unsigned *seed,int key_field ,int batchsize);
std::vector<RequestWrapper<unsigned long long, data_t *>> generatePopulation(ifstream &inFile, unsigned *seed,int key_field ,int batchsize);

struct ServerConf {
    int threads;
    int cpu_threads;

    int gpus;
    int streams;
    std::string modelFile;
    bool train;
    int size;
    int batchSize;
    bool cache;

    ServerConf() {
        batchSize = BATCHSIZE;
        modelFile = "";
        cpu_threads = NUM_THREADS;
        threads = 2;//1;//4;
        gpus = 1;
        streams = 10;//10;
        size = 1000000;
        train = false;
        cache = true;
    }

    explicit ServerConf(const std::string &filename) {
        pt::ptree root;
        pt::read_json(filename, root);
        cpu_threads = root.get<int>("cpu_threads", NUM_THREADS);
        threads = root.get<int>("threads", 4);
        streams = root.get<int>("streams", 2);
        gpus = root.get<int>("gpus", 2);
        modelFile = root.get<std::string>("modelFile", "");
        train = root.get<bool>("train", false);
        size = root.get<int>("size", 1000000);
        batchSize = root.get<int>("batchSize", BATCHSIZE);
        cache = root.get<bool>("cache", true);
    }

    void persist(const std::string &filename) const {
        pt::ptree root;
        root.put("threads", threads);
        root.put("streams", streams);
        root.put("gpus", gpus);
        root.put("modelFile", modelFile);
        root.put("train", train);
        root.put("size", size);
        root.put("batchSize", batchSize);
        root.put("cache", cache);
        pt::write_json(filename, root);
    }

    ~ServerConf() = default;

};

int main(int argc, char **argv) {

    ServerConf sconf;
    ifstream inFile("cluster001.csv", ios::in);

    int workload_count = count_workload(inFile);
    inFile.close();

    inFile.open("cluster001.csv");
    float ratio_train = 0.01f, 
           ratio_population = 0.3f;


    std::vector<PartitionedSlabUnifiedConfig> conf;
    for (int i = 0; i < sconf.gpus; i++) {
        for (int j = 0; j < sconf.streams; j++) {
            gpuErrchk(hipSetDevice(i));
            hipStream_t stream = hipStreamDefault;
            if (j != 0) {
                gpuErrchk(hipStreamCreate(&stream));
            }
            conf.push_back({sconf.size, i, stream});
        }
    }


    std::unique_ptr<KVStoreCtx<Model>> ctx = nullptr;
    unsigned tseed = time(nullptr);
    std::vector<std::pair<unsigned long long, unsigned>> trainVec;
    std::hash<unsigned long long> hfn{};

    int current_workload = workload_count * ratio_train;
    workload_count -= current_workload;
    // for (int i = 0; i < 50; i++) {
    //     BatchWrapper b = generatePopulation(inFile,&tseed, 1, );
    //     for (auto &elm : b) {
    //         trainVec.push_back({elm.key, hfn(elm.key)});
    //     }
        
    // }

   
    while (current_workload > 0){
        BatchWrapper b = generatePopulation(inFile,&tseed, 1,current_workload >= sconf.batchSize ? sconf.batchSize : current_workload);
        for (auto &elm : b) {
            trainVec.push_back({elm.key, hfn(elm.key)});
        }
        current_workload -= sconf.batchSize;
    }
    

        Model m;
        m.train(trainVec);
    //     m.persist("./temp.json");
        ctx = std::make_unique<KVStoreCtx<Model>>(conf, sconf.cpu_threads, m);
    // }

    GeneralClient<Model> *client = nullptr;
    if (sconf.cache) {
        if (sconf.gpus == 0) {
            client = new JustCacheKVStoreClient<Model>(*ctx);
        } else {
            client = new KVStoreClient<Model>(*ctx);
        }
    } else {
        client = new NoCacheKVStoreClient<Model>(*ctx);
    }
    
    init_loadbalance(sconf.cpu_threads);
    std::vector<BatchWrapper> work;
    std::vector<BatchWrapper> population;

    current_workload = workload_count * ratio_population;
    workload_count -= current_workload;

    while (current_workload > 0){
        population.push_back(generatePopulation(inFile, &tseed, 1, current_workload >= sconf.batchSize ? sconf.batchSize : current_workload));
        current_workload -= sconf.batchSize;
        cout << current_workload << endl;
    }
    // for(int i = 0; i < 1000; i++)
    //     population.push_back(generatePopulation(inFile, &tseed, 1, 512));


    for(auto i : population)
    {
        auto rb = std::make_shared<LocalCommunication>(i.size());
        auto start = std::chrono::high_resolution_clock::now();
        client->batch(i, rb, start);
    }

    while (workload_count > 0){
        work.push_back(generatePopulation(inFile, &tseed, 1, workload_count >= sconf.batchSize ? sconf.batchSize : workload_count));
        workload_count -= sconf.batchSize;
    }

  
    // for(int i = 0; i < 1000; i++)
    //     work.push_back(generateWorkloadBatch(inFile, &tseed, 1, 512));

    auto startTime = std::chrono::high_resolution_clock::now();

    std::cout << "before test" << std::endl;

    for(auto i : work)
    {
        auto rb = std::make_shared<LocalCommunication>(i.size());
        auto start = std::chrono::high_resolution_clock::now();
        client->batch(i, rb, start);
    }


    
    auto endTimeArrival = std::chrono::high_resolution_clock::now();

    auto endTime = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> dur = endTime - startTime;
    std::chrono::duration<double> durArr = endTimeArrival - startTime;
    size_t ops = client->getOps();
    // std::cerr << "Throughput (ops) " << ((double) ops)  << std::endl;

    std::cerr << "Hits rate: " << client->hitRate() << std::endl;
    // client->stat();
    delete client;
    // generateWorkloadBatch(inFile, &tseed, 1, 10);
    // generateWorkloadBatch(inFile, &tseed, 1, 10);
    inFile.close();
    return 0;
}

void usage(char *command) {
    using namespace std;
    cout << command << " [-f <config file>]" << std::endl;
}

int count_workload(ifstream &inFile){
    int counter = 0;
    string line;
    while (getline(inFile,line)){ 
        counter++;
    }

    return counter;
}
std::vector<RequestWrapper<unsigned long long, data_t *>> generateWorkloadBatch(ifstream &inFile, unsigned *seed,int key_field ,int batchsize) {
    std::vector<RequestWrapper<unsigned long long, data_t *>> vec;
    
    int counter = 0;
    string line;
    while (getline(inFile,line) && counter < batchsize){ 
        string field;
        string substr;
        istringstream readstr(line);

        unsigned long long l = 0;
        unsigned int type = REQUEST_GET;

        for(int j = 0;j < key_field+1;j++){ 
            getline(readstr,field,','); 
            if (j == key_field) {
            	// substr = string {field.c_str(), field.c_str().length()-8,field.c_str().length()};
                substr = field.substr(field.length()-8,field.length());
            	 for (int i = 0; i < 8; ++i) {
    			    l = l | ((unsigned long long)substr[i] << (8 * i));
  		        }
                //   cout << l << endl;

                if (rand_r(seed) % 100 < RATIO_OF_READS) {
                    type = REQUEST_GET;
                    vec.push_back({l, 0, nullptr, type});
                }
                else 
                    if (rand_r(seed) % 100 < 50) {
                        type = REQUEST_INSERT;
                        vec.push_back({l, 0,new data_t(KEY_SIZE), type});
                    } else {
                        type = REQUEST_REMOVE;
                        vec.push_back({l, 0, nullptr, type});
                    }
        
            }
        }

        counter++;

    }

    return vec;
    
}

std::vector<RequestWrapper<unsigned long long, data_t *>> generatePopulation(ifstream &inFile, unsigned *seed,int key_field ,int batchsize) {
    std::vector<RequestWrapper<unsigned long long, data_t *>> vec;
    
    int counter = 0;
    string line;
    while (getline(inFile,line) && counter < batchsize){ 
        string field;
        string substr;
        istringstream readstr(line);

        unsigned long long l = 0;
        unsigned int type = REQUEST_GET;

        for(int j = 0;j < key_field+1;j++){ 
            getline(readstr,field,','); 
            if (j == key_field) {
            	// substr = string {field.c_str(), field.c_str().length()-8,field.c_str().length()};
                substr = field.substr(field.length()-8,field.length());
            	 for (int i = 0; i < 8; ++i) {
    			    l = l | ((unsigned long long)substr[i] << (8 * i));
  		        }
                //   cout << l << endl;



            type = REQUEST_INSERT;
            vec.push_back({l, 0,new data_t(KEY_SIZE), type});

                       
                    
        
            }
        }

        counter++;

    }

    return vec;
    
}

std::vector<BatchWrapper> getPopulationBatches(ifstream &inFile, unsigned int *seed,int key_field ,unsigned int nums) {
    std::vector<BatchWrapper> batches;
    for(int i=0; i< nums; i++){
        std::vector<RequestWrapper<unsigned long long, data_t *>> vec;
        int counter = 0;
        string line;
        string field;
        while (getline(inFile,line) && counter < nums){ 
            string number;
            string substr;
            istringstream readstr(line);
    
            unsigned long long l = 0;
    
            for(int j = 0;j < key_field+1;j++){ 
                getline(readstr,number,','); 
                if (j == key_field) {
                    substr = string {number.c_str(), 0,8};
                     for (int i = 0; i < 8; ++i) {
                        l = l | ((unsigned long long)substr[i] << (8 * i));
                      }
    
                    
                    vec.push_back({l, 0,new data_t(KEY_SIZE), REQUEST_INSERT});
            
                }
                cout << l << endl;
            }
    
            counter++;
    
        }
        batches.push_back(vec);
    }

    return batches;
   
}
